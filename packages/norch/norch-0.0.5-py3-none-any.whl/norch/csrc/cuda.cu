#include "hip/hip_runtime.h"
#include "tensor.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>

#define THREADS_PER_BLOCK 128
#define TILE_SIZE 32

__host__ void cpu_to_cuda(Tensor* tensor, int device_id) {

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (device_id >= deviceCount) {
        fprintf(stderr, "Could not send tensor to device %d, only %d devices available\n", device_id, deviceCount);
        exit(1);
    }
    
    hipSetDevice(device_id); 

    float* data_tmp;
    hipMalloc((void **)&data_tmp, tensor->size * sizeof(float));
    hipMemcpy(data_tmp, tensor->data, tensor->size * sizeof(float), hipMemcpyHostToDevice);

    tensor->data = data_tmp;

    const char* device_str = "cuda";
    tensor->device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor->device, device_str); 
}

__host__ void cuda_to_cpu(Tensor* tensor) {
    float* data_tmp = (float*)malloc(tensor->size * sizeof(float));

    hipMemcpy(data_tmp, tensor->data, tensor->size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(tensor->data);

    tensor->data = data_tmp;

    const char* device_str = "cpu";
    tensor->device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor->device, device_str); 
}

__global__ void add_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] + data2[i];
    }
}

__host__ void add_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void add_broadcasted_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int* broadcasted_shape, int* strides1, int*strides2, int max_ndim, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    
    int index1 = 0, index2 = 0;
    int linear_index = i;
    for (int j = max_ndim - 1; j >= 0; j--) {
        int pos = linear_index % broadcasted_shape[j];
        linear_index /= broadcasted_shape[j];
        if (strides1[j] != 0) index1 += pos * strides1[j];
        if (strides2[j] != 0) index2 += pos * strides2[j];
    }
    result_data[i] = data1[index1] + data2[index2];
}

__host__ void add_broadcasted_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data, int* broadcasted_shape, int broadcasted_size) {
    int max_ndim = tensor1->ndim > tensor2->ndim ? tensor1->ndim : tensor2->ndim;

    int* strides1 = (int*)malloc(max_ndim * sizeof(int));
    int* strides2 = (int*)malloc(max_ndim * sizeof(int));
    if (strides1 == NULL || strides2 == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(1);
    }

    int stride1 = 1, stride2 = 1;
    for (int i = max_ndim - 1; i >= 0; i--) {
        int dim1 = i < tensor1->ndim ? tensor1->shape[tensor1->ndim - max_ndim + i] : 1;
        int dim2 = i < tensor2->ndim ? tensor2->shape[tensor2->ndim - max_ndim + i] : 1;
        strides1[i] = dim1 == broadcasted_shape[i] ? stride1 : 0;
        strides2[i] = dim2 == broadcasted_shape[i] ? stride2 : 0;
        stride1 *= (dim1 == broadcasted_shape[i]) ? dim1 : 1;
        stride2 *= (dim2 == broadcasted_shape[i]) ? dim2 : 1;
    }
    
    int* d_broadcasted_shape;
    int* d_strides1;
    int* d_strides2;

    hipMalloc((void**)&d_broadcasted_shape, max_ndim * sizeof(int));
    hipMemcpy(d_broadcasted_shape, broadcasted_shape, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides1, max_ndim * sizeof(int));
    hipMemcpy(d_strides1, strides1, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides2, max_ndim * sizeof(int));
    hipMemcpy(d_strides2, strides2, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    int number_of_blocks = (broadcasted_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_broadcasted_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, d_broadcasted_shape, d_strides1, d_strides2, max_ndim, broadcasted_size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
    hipFree(d_broadcasted_shape);
}

__global__ void sum_tensor_cuda_kernel(float* data, float* result_data, int size) {
    __shared__ float partial_sum[THREADS_PER_BLOCK * sizeof(float)];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    partial_sum[tid] = (i < size) ? data[i] : 0;

    __syncthreads();

    // Perform block-wise reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_sum[tid] += partial_sum[tid + s];
        }
        __syncthreads();
    }

    // Write block sum to global memory
    if (tid == 0) {
        result_data[blockIdx.x] = partial_sum[0];
    }
}

__global__ void sum_tensor_cuda_kernel_axis(float* data, float* result_data, int* strides, int* shape, int axis, int ndim, int axis_stride, int size, int result_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < result_size) {
        for (int i = 0; i < shape[axis]; i++) {
            int index = 0;
            int remainder = tid;
            for (int k = ndim - 2; k >= 0; k--) {
                index += (remainder % shape[k < axis ? k : k + 1]) * strides[k < axis ? k : k + 1];
                remainder /= shape[k < axis ? k : k + 1];
            }
            index += i * axis_stride;

            atomicAdd(&result_data[tid], data[index]);
        }
    }
}


__host__ void sum_tensor_cuda(Tensor* tensor, float* result_data, int axis) {

    if (axis == -1) {
        hipMemcpy(result_data, tensor->data, tensor->size * sizeof(float), hipMemcpyHostToDevice);
        
        int num_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // First-level reduction
        sum_tensor_cuda_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

        // If necessary, perform multiple levels of reduction
        while (num_blocks > 1) {
            int num_blocks_next = (num_blocks + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            sum_tensor_cuda_kernel<<<num_blocks_next, THREADS_PER_BLOCK>>>(result_data, result_data, num_blocks);
            num_blocks = num_blocks_next;
        }

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();
        
    } else {
        int axis_stride = tensor->strides[axis];

        // Calculate the size of the resulting tensor
        int result_size = 1;
        for (int i = 0; i < tensor->ndim; i++) {
            if (i != axis) {
                result_size *= tensor->shape[i];
            }
        }

        // Allocate memory for strides and shape on the device
        int* d_strides;
        int* d_shape;
        hipMalloc(&d_strides, tensor->ndim * sizeof(int));
        hipMalloc(&d_shape, tensor->ndim * sizeof(int));
        hipMemcpy(d_strides, tensor->strides, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_shape, tensor->shape, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);

        // Initialize result_data to 0
        hipMemset(result_data, 0, result_size * sizeof(float));

        int num_threads = result_size;
        int num_blocks = (num_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        sum_tensor_cuda_kernel_axis<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, d_strides, d_shape, axis, tensor->ndim, axis_stride, tensor->size, result_size);

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();

        // Free allocated memory
        hipFree(d_strides);
        hipFree(d_shape);
    }
}

__global__ void max_tensor_cuda_kernel(float* data, float* result_data, int size) {
    __shared__ float partial_max[THREADS_PER_BLOCK * sizeof(float)];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    partial_max[tid] = (i < size) ? data[i] : -FLT_MAX;

    __syncthreads();

    // Perform block-wise reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_max[tid] = fmax(partial_max[tid], partial_max[tid + s]);
        }
        __syncthreads();
    }

    // Write block sum to global memory
    if (tid == 0) {
        result_data[blockIdx.x] = partial_max[0];
    }
}

__device__ float atomicMaxFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
                        __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old);
}

__global__ void max_tensor_cuda_kernel_axis(float* data, float* result_data, int* strides, int* shape, int axis, int ndim, int axis_stride, int size, int result_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < result_size) {
        for (int i = 0; i < shape[axis]; i++) {
            int index = 0;
            int remainder = tid;
            for (int k = ndim - 2; k >= 0; k--) {
                index += (remainder % shape[k < axis ? k : k + 1]) * strides[k < axis ? k : k + 1];
                remainder /= shape[k < axis ? k : k + 1];
            }
            index += i * axis_stride;

            atomicMaxFloat(&result_data[tid], data[index]);
        }
    }
}

__host__ void max_tensor_cuda(Tensor* tensor, float* result_data, int axis) {

    if (axis == -1) {
        hipMemcpy(result_data, tensor->data, tensor->size * sizeof(float), hipMemcpyHostToDevice);
        
        int num_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // First-level reduction
        max_tensor_cuda_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

        // If necessary, perform multiple levels of reduction
        while (num_blocks > 1) {
            int num_blocks_next = (num_blocks + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            max_tensor_cuda_kernel<<<num_blocks_next, THREADS_PER_BLOCK>>>(result_data, result_data, num_blocks);
            num_blocks = num_blocks_next;
        }

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();
        
    } else {
        int axis_stride = tensor->strides[axis];

        // Calculate the size of the resulting tensor
        int result_size = 1;
        for (int i = 0; i < tensor->ndim; i++) {
            if (i != axis) {
                result_size *= tensor->shape[i];
            }
        }

        // Allocate memory for strides and shape on the device
        int* d_strides;
        int* d_shape;
        hipMalloc(&d_strides, tensor->ndim * sizeof(int));
        hipMalloc(&d_shape, tensor->ndim * sizeof(int));
        hipMemcpy(d_strides, tensor->strides, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_shape, tensor->shape, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);

        // Initialize result_data to 0
        float neg_inf = -FLT_MAX;
        hipMemset(result_data, *reinterpret_cast<int*>(&neg_inf), result_size * sizeof(float));

        int num_threads = result_size;
        int num_blocks = (num_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        max_tensor_cuda_kernel_axis<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, d_strides, d_shape, axis, tensor->ndim, axis_stride, tensor->size, result_size);

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();

        // Free allocated memory
        hipFree(d_strides);
        hipFree(d_shape);
    }
}

__global__ void min_tensor_cuda_kernel(float* data, float* result_data, int size) {
    __shared__ float partial_min[THREADS_PER_BLOCK * sizeof(float)];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    partial_min[tid] = (i < size) ? data[i] : FLT_MAX;

    __syncthreads();

    // Perform block-wise reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_min[tid] = fmin(partial_min[tid], partial_min[tid + s]);
        }
        __syncthreads();
    }

    // Write block sum to global memory
    if (tid == 0) {
        result_data[blockIdx.x] = partial_min[0];
    }
}

__device__ float atomicMinFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
                        __float_as_int(fminf(val, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old);
}

__global__ void min_tensor_cuda_kernel_axis(float* data, float* result_data, int* strides, int* shape, int axis, int ndim, int axis_stride, int size, int result_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < result_size) {
        for (int i = 0; i < shape[axis]; i++) {
            int index = 0;
            int remainder = tid;
            for (int k = ndim - 2; k >= 0; k--) {
                index += (remainder % shape[k < axis ? k : k + 1]) * strides[k < axis ? k : k + 1];
                remainder /= shape[k < axis ? k : k + 1];
            }
            index += i * axis_stride;

            atomicMinFloat(&result_data[tid], data[index]);
        }
    }
}

__host__ void min_tensor_cuda(Tensor* tensor, float* result_data, int axis) {

    if (axis == -1) {
        hipMemcpy(result_data, tensor->data, tensor->size * sizeof(float), hipMemcpyHostToDevice);
        
        int num_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // First-level reduction
        min_tensor_cuda_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

        // If necessary, perform multiple levels of reduction
        while (num_blocks > 1) {
            int num_blocks_next = (num_blocks + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            min_tensor_cuda_kernel<<<num_blocks_next, THREADS_PER_BLOCK>>>(result_data, result_data, num_blocks);
            num_blocks = num_blocks_next;
        }

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();
        
    } else {
        int axis_stride = tensor->strides[axis];

        // Calculate the size of the resulting tensor
        int result_size = 1;
        for (int i = 0; i < tensor->ndim; i++) {
            if (i != axis) {
                result_size *= tensor->shape[i];
            }
        }

        // Allocate memory for strides and shape on the device
        int* d_strides;
        int* d_shape;
        hipMalloc(&d_strides, tensor->ndim * sizeof(int));
        hipMalloc(&d_shape, tensor->ndim * sizeof(int));
        hipMemcpy(d_strides, tensor->strides, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_shape, tensor->shape, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);

        // Initialize result_data to 0
        float inf = FLT_MAX;
        hipMemset(result_data, *reinterpret_cast<int*>(&inf), result_size * sizeof(float));

        int num_threads = result_size;
        int num_blocks = (num_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        min_tensor_cuda_kernel_axis<<<num_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, d_strides, d_shape, axis, tensor->ndim, axis_stride, tensor->size, result_size);

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }

        hipDeviceSynchronize();

        // Free allocated memory
        hipFree(d_strides);
        hipFree(d_shape);
    }
}



__global__ void sub_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] - data2[i];
    }
}

__host__ void sub_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sub_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void sub_broadcasted_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int* broadcasted_shape, int* strides1, int*strides2, int max_ndim, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    
    int index1 = 0, index2 = 0;
    int linear_index = i;
    for (int j = max_ndim - 1; j >= 0; j--) {
        int pos = linear_index % broadcasted_shape[j];
        linear_index /= broadcasted_shape[j];
        if (strides1[j] != 0) index1 += pos * strides1[j];
        if (strides2[j] != 0) index2 += pos * strides2[j];
    }
    result_data[i] = data1[index1] - data2[index2];
}

__host__ void sub_broadcasted_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data, int* broadcasted_shape, int broadcasted_size) {
    int max_ndim = tensor1->ndim > tensor2->ndim ? tensor1->ndim : tensor2->ndim;

    int* strides1 = (int*)malloc(max_ndim * sizeof(int));
    int* strides2 = (int*)malloc(max_ndim * sizeof(int));
    if (strides1 == NULL || strides2 == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(1);
    }

    int stride1 = 1, stride2 = 1;
    for (int i = max_ndim - 1; i >= 0; i--) {
        int dim1 = i < tensor1->ndim ? tensor1->shape[tensor1->ndim - max_ndim + i] : 1;
        int dim2 = i < tensor2->ndim ? tensor2->shape[tensor2->ndim - max_ndim + i] : 1;
        strides1[i] = dim1 == broadcasted_shape[i] ? stride1 : 0;
        strides2[i] = dim2 == broadcasted_shape[i] ? stride2 : 0;
        stride1 *= (dim1 == broadcasted_shape[i]) ? dim1 : 1;
        stride2 *= (dim2 == broadcasted_shape[i]) ? dim2 : 1;
    }
    
    int* d_broadcasted_shape;
    int* d_strides1;
    int* d_strides2;

    hipMalloc((void**)&d_broadcasted_shape, max_ndim * sizeof(int));
    hipMemcpy(d_broadcasted_shape, broadcasted_shape, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides1, max_ndim * sizeof(int));
    hipMemcpy(d_strides1, strides1, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides2, max_ndim * sizeof(int));
    hipMemcpy(d_strides2, strides2, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    int number_of_blocks = (broadcasted_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sub_broadcasted_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, d_broadcasted_shape, d_strides1, d_strides2, max_ndim, broadcasted_size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
    hipFree(d_broadcasted_shape);
}

__global__ void elementwise_mul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] * data2[i];
    }
}

__host__ void elementwise_mul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    elementwise_mul_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void scalar_mul_tensor_cuda_kernel(float* data, float scalar, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = scalar * data[i];
    }
}

__host__ void scalar_mul_tensor_cuda(Tensor* tensor, float scalar, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    scalar_mul_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, scalar, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void scalar_div_tensor_cuda_kernel(float scalar, float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = scalar / data[i];
    }
}

__host__ void scalar_div_tensor_cuda(float scalar, Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    scalar_div_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(scalar, tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void tensor_div_scalar_cuda_kernel(float* data, float scalar, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data[i] / scalar;
    }
}

__host__ void tensor_div_scalar_cuda(Tensor* tensor, float scalar, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    tensor_div_scalar_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, scalar, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void tensor_div_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] / data2[i];
    }
}

__host__ void tensor_div_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    tensor_div_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

/*__global__ void matmul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int rows1, int cols1, int cols2) {    

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows1 && col < cols2) {
        float sum = 0.0;
        for (int k = 0; k < cols1; k++) {
            sum += data1[row * cols1 + k] * data2[k * cols2 + col];
        }
        result_data[row * cols2 + col] = sum;
    }

}*/

__global__ void matmul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int rows1, int cols1, int cols2) {    

    // Shared memory for tiles
    __shared__ float tile1[TILE_SIZE][TILE_SIZE];
    __shared__ float tile2[TILE_SIZE][TILE_SIZE];

    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Output position
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0;

    // Iterate over tiles
    for (int i = 0; i < (cols1 + TILE_SIZE - 1) / TILE_SIZE; ++i) {

        // Load tiles into shared memory
        if (row < rows1 && i * TILE_SIZE + tx < cols1)
            tile1[ty][tx] = data1[row * cols1 + i * TILE_SIZE + tx];
        else
            tile1[ty][tx] = 0.0;

        if (col < cols2 && i * TILE_SIZE + ty < cols1)
            tile2[ty][tx] = data2[(i * TILE_SIZE + ty) * cols2 + col];
        else
            tile2[ty][tx] = 0.0;

        // Synchronize threads
        __syncthreads();

        // Accumulate sum
        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tile1[ty][k] * tile2[k][tx];

        // Synchronize threads
        __syncthreads();
    }

    // Write result to global memory
    if (row < rows1 && col < cols2)
        result_data[row * cols2 + col] = sum;
}


__host__ void matmul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int rows1 = tensor1->shape[0];
    int cols1 = tensor1->shape[1];
    int cols2 = tensor2->shape[1];

    dim3 threadsPerBlock(16, 16);
    dim3 number_of_blocks((cols2 + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmul_tensor_cuda_kernel<<<number_of_blocks, threadsPerBlock>>>(tensor1->data, tensor2->data, result_data, rows1, cols1, cols2);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void batched_matmul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int batch_size, int rows1, int cols1, int cols2) {
    int batch = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows1 && col < cols2) {
        float sum = 0.0f;
        for (int k = 0; k < cols1; ++k) {
            sum += data1[batch * rows1 * cols1 + row * cols1 + k] * 
                   data2[batch * cols1 * cols2 + k * cols2 + col];
        }
        result_data[batch * rows1 * cols2 + row * cols2 + col] = sum;
    }    
}

__host__ void batched_matmul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {

    int batch_size = tensor2->shape[0];
    int rows1 = tensor1->shape[1];
    int cols1 = tensor1->shape[2];
    int cols2 = tensor2->shape[2];

    dim3 threadsPerBlock(16, 16);
    dim3 number_of_blocks((cols2 + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows1 + threadsPerBlock.y - 1) / threadsPerBlock.y, batch_size);
    batched_matmul_tensor_cuda_kernel<<<number_of_blocks, threadsPerBlock>>>(tensor1->data, tensor2->data, result_data, batch_size, rows1, cols1, cols2);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void broadcasted_batched_matmul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int batch_size, int rows1, int cols1, int cols2) {
    int batch = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows1 && col < cols2) {
        float sum = 0.0f;
        for (int k = 0; k < cols1; ++k) {
            sum += data1[row * cols1 + k] * 
                   data2[batch * cols1 * cols2 + k * cols2 + col];
        }
        result_data[batch * rows1 * cols2 + row * cols2 + col] = sum;
    }    
}

__host__ void broadcasted_batched_matmul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int batch_size = tensor2->shape[0];
    int rows1 = tensor1->shape[0];
    int cols1 = tensor1->shape[1];
    int cols2 = tensor2->shape[2];

    dim3 threadsPerBlock(16, 16);
    dim3 number_of_blocks((cols2 + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows1 + threadsPerBlock.y - 1) / threadsPerBlock.y, batch_size);
    broadcasted_batched_matmul_tensor_cuda_kernel<<<number_of_blocks, threadsPerBlock>>>(tensor1->data, tensor2->data, result_data, batch_size, rows1, cols1, cols2);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void tensor_pow_scalar_cuda_kernel(float* data, float exponent, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = powf(data[i], exponent);
    }
}

__host__ void tensor_pow_scalar_cuda(Tensor* tensor, float exponent, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    tensor_pow_scalar_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, exponent, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void scalar_pow_tensor_cuda_kernel(float base, float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = powf(base, data[i]);
    }
}

__host__ void scalar_pow_tensor_cuda(float base, Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    scalar_pow_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(base, tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void log_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = logf(data[i]);
    }
}

__host__ void log_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    log_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void equal_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = (data1[i] == data2[i]) ? 1.0f : 0.0f;
    }
}

__host__ void equal_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    equal_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void equal_broadcasted_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int* broadcasted_shape, int* strides1, int*strides2, int max_ndim, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    
    int index1 = 0, index2 = 0;
    int linear_index = i;
    for (int j = max_ndim - 1; j >= 0; j--) {
        int pos = linear_index % broadcasted_shape[j];
        linear_index /= broadcasted_shape[j];
        if (strides1[j] != 0) index1 += pos * strides1[j];
        if (strides2[j] != 0) index2 += pos * strides2[j];
    }
    result_data[i] = (data1[index1] == data2[index2]) ? 1.0f : 0.0f;
}

__host__ void equal_broadcasted_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data, int* broadcasted_shape, int broadcasted_size) {
    int max_ndim = tensor1->ndim > tensor2->ndim ? tensor1->ndim : tensor2->ndim;

    int* strides1 = (int*)malloc(max_ndim * sizeof(int));
    int* strides2 = (int*)malloc(max_ndim * sizeof(int));
    if (strides1 == NULL || strides2 == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(1);
    }

    int stride1 = 1, stride2 = 1;
    for (int i = max_ndim - 1; i >= 0; i--) {
        int dim1 = i < tensor1->ndim ? tensor1->shape[tensor1->ndim - max_ndim + i] : 1;
        int dim2 = i < tensor2->ndim ? tensor2->shape[tensor2->ndim - max_ndim + i] : 1;
        strides1[i] = dim1 == broadcasted_shape[i] ? stride1 : 0;
        strides2[i] = dim2 == broadcasted_shape[i] ? stride2 : 0;
        stride1 *= (dim1 == broadcasted_shape[i]) ? dim1 : 1;
        stride2 *= (dim2 == broadcasted_shape[i]) ? dim2 : 1;
    }
    
    int* d_broadcasted_shape;
    int* d_strides1;
    int* d_strides2;

    hipMalloc((void**)&d_broadcasted_shape, max_ndim * sizeof(int));
    hipMemcpy(d_broadcasted_shape, broadcasted_shape, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides1, max_ndim * sizeof(int));
    hipMemcpy(d_strides1, strides1, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_strides2, max_ndim * sizeof(int));
    hipMemcpy(d_strides2, strides2, max_ndim * sizeof(int), hipMemcpyHostToDevice);

    int number_of_blocks = (broadcasted_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    equal_broadcasted_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, d_broadcasted_shape, d_strides1, d_strides2, max_ndim, broadcasted_size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
    hipFree(d_broadcasted_shape);
}


__global__ void ones_like_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = 1.0;
    }
}

__host__ void ones_like_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    ones_like_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void zeros_like_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = 0.0;
    }
}

__host__ void zeros_like_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    zeros_like_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}


__global__ void transpose_1D_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data[i];
    }
}

__host__ void transpose_1D_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    transpose_1D_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void transpose_2D_tensor_cuda_kernel(float* data, float* result_data, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        result_data[j * rows + i] = data[i * cols + j];
    }
}

__host__ void transpose_2D_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int rows = tensor->shape[0];
    int cols = tensor->shape[1];

    dim3 threadsPerBlock(16, 16);
    dim3 number_of_blocks((rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (cols + threadsPerBlock.y - 1) / threadsPerBlock.y);
    transpose_2D_tensor_cuda_kernel<<<number_of_blocks, threadsPerBlock>>>(tensor->data, result_data, rows, cols);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void transpose_3D_tensor_cuda_kernel(float* data, float* result_data, int batch, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < batch && j < rows && k < cols) {
        result_data[k * rows * batch + j * batch + i] = data[i * rows * cols + j * cols + k];
    }
}

__host__ void transpose_3D_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int batch = tensor->shape[0];
    int rows = tensor->shape[1];
    int cols = tensor->shape[2];

    dim3 threadsPerBlock(8, 8, 8);
    dim3 number_of_blocks((batch + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y, (cols + threadsPerBlock.z - 1) / threadsPerBlock.z);
    transpose_3D_tensor_cuda_kernel<<<number_of_blocks, threadsPerBlock>>>(tensor->data, result_data, batch, rows, cols);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}


__global__ void assign_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data[i];
    }
}

__host__ void assign_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    assign_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void make_contiguous_tensor_cuda_kernel(float* data, float* result_data, int ndim, int size, int* strides, int* new_strides) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int index = 0;
        int offset = i;
        for (int j = 0; j < ndim; j++) {
            index += (offset / new_strides[j]) * strides[j];
            offset %= new_strides[j];
        }
        result_data[i] = data[index];
    }
}

__host__ void make_contiguous_tensor_cuda(Tensor* tensor, float* result_data, int* new_strides) {
    
    int* d_strides;
    hipMalloc((void **)&d_strides, tensor->ndim * sizeof(int));
    hipMemcpy(d_strides, tensor->strides, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);
    
    int* d_new_strides;
    hipMalloc((void **)&d_new_strides, tensor->ndim * sizeof(int));
    hipMemcpy(d_new_strides, new_strides, tensor->ndim * sizeof(int), hipMemcpyHostToDevice);

    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    make_contiguous_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->ndim, tensor->size, d_strides, d_new_strides);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();

    // Free old data and update tensor properties
    hipFree(tensor->data);
    free(tensor->strides);
    tensor->data = result_data;
    tensor->strides = new_strides;
}

__global__ void sin_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = sinf(data[i]);
    }
}

__host__ void sin_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sin_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void cos_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = cosf(data[i]);
    }
}

__host__ void cos_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cos_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void sigmoid_tensor_cuda_kernel(float* data, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // avoid overflow
        if (data[i] >= 0) {

            float z = expf(-data[i]);
            result_data[i] = 1 / (1 + z);

        } else {

            float z = expf(data[i]);
            result_data[i] = z / (1 + z);
        }
    }
}

__host__ void sigmoid_tensor_cuda(Tensor* tensor, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sigmoid_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}




